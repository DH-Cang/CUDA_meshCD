//**************************************************************************************
//  Copyright (C) 2022 - 2024, Min Tang (tang_m@zju.edu.cn)
//  All rights reserved.
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions
//  are met:
//     1. Redistributions of source code must retain the above copyright
//        notice, this list of conditions and the following disclaimer.
//     2. Redistributions in binary form must reproduce the above copyright
//        notice, this list of conditions and the following disclaimer in the
//        documentation and/or other materials provided with the distribution.
//     3. The names of its contributors may not be used to endorse or promote
//        products derived from this software without specific prior written
//        permission.
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
//  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
//  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
//  A PARTICULAR PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
//  CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
//  EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
//  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
//  PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
//  LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
//	NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
//	SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//**************************************************************************************

#include <set>
#include <iostream>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>

using namespace std;
#include "mat3f.h"
#include "box.h"
#include "crigid.cuh"
#include "cuda_colide.cuh"
#include "book.h"

#define GPU_ACCELE


#ifndef GPU_ACCELE
// very robust triangle intersection test
// uses no divisions
// works on coplanar triangles

bool
triContact(vec3f& P1, vec3f& P2, vec3f& P3, vec3f& Q1, vec3f& Q2, vec3f& Q3)
{
	vec3f p1;
	vec3f p2 = P2 - P1;
	vec3f p3 = P3 - P1;
	vec3f q1 = Q1 - P1;
	vec3f q2 = Q2 - P1;
	vec3f q3 = Q3 - P1;

	vec3f e1 = p2 - p1;
	vec3f e2 = p3 - p2;
	vec3f e3 = p1 - p3;

	vec3f f1 = q2 - q1;
	vec3f f2 = q3 - q2;
	vec3f f3 = q1 - q3;

	vec3f n1 = e1.cross(e2);
	vec3f m1 = f1.cross(f2);

	vec3f g1 = e1.cross(n1);
	vec3f g2 = e2.cross(n1);
	vec3f g3 = e3.cross(n1);

	vec3f  h1 = f1.cross(m1);
	vec3f h2 = f2.cross(m1);
	vec3f h3 = f3.cross(m1);

	vec3f ef11 = e1.cross(f1);
	vec3f ef12 = e1.cross(f2);
	vec3f ef13 = e1.cross(f3);
	vec3f ef21 = e2.cross(f1);
	vec3f ef22 = e2.cross(f2);
	vec3f ef23 = e2.cross(f3);
	vec3f ef31 = e3.cross(f1);
	vec3f ef32 = e3.cross(f2);
	vec3f ef33 = e3.cross(f3);

	// now begin the series of tests
	if (!project3(n1, q1, q2, q3)) return false;
	if (!project3(m1, -q1, p2 - q1, p3 - q1)) return false;

	if (!project6(ef11, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(ef12, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(ef13, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(ef21, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(ef22, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(ef23, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(ef31, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(ef32, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(ef33, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(g1, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(g2, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(g3, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(h1, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(h2, p1, p2, p3, q1, q2, q3)) return false;
	if (!project6(h3, p1, p2, p3, q1, q2, q3)) return false;

	return true;
}
#endif


// input: kmesh0(this) with transformation t0
//        kmesh1(other) with transformation t1
void
kmesh::collide(const kmesh* other, const transf& t0, const transf &t1, std::vector<id_pair>& rets)
{

#ifndef GPU_ACCELE

	for (int i = 0; i < _num_tri; i++) {
		//printf("checking %d of %d...\n", i, _num_tri);

		for (int j = 0; j < other->_num_tri; j++) {
			vec3f v0, v1, v2;
			this->getTriangleVtxs(i, v0, v1, v2);
			vec3f p0 = t0.getVertex(v0);
			vec3f p1 = t0.getVertex(v1);
			vec3f p2 = t0.getVertex(v2);

			other->getTriangleVtxs(j, v0, v1, v2);
			vec3f q0 = t1.getVertex(v0);
			vec3f q1 = t1.getVertex(v1);
			vec3f q2 = t1.getVertex(v2);

			if (triContact(p0, p1, p2, q0, q1, q2))
				rets.push_back(id_pair(i, j, false));
		}
}
#else
	thrust::device_vector<tri3f> d_mesh0_tris(_tris, _tris + _num_tri);
	thrust::device_vector<tri3f> d_mesh1_tris(other->_tris, other->_tris + other->_num_tri);
	thrust::device_vector<vec3f> d_mesh0_vtxs(_vtxs, _vtxs + _num_vtx);
	thrust::device_vector<vec3f> d_mesh1_vtxs(other->_vtxs, other->_vtxs + other->_num_vtx);
	thrust::device_vector<bool> d_triangle0_result(_num_tri);
	thrust::device_vector<bool> d_triangle1_result(other->_num_tri);

	transf* d_transform0;
	transf* d_transform1;
	// allocate memory
	HANDLE_ERROR(hipMalloc((void**)&d_transform0, sizeof(transf)));
	HANDLE_ERROR(hipMalloc((void**)&d_transform1, sizeof(transf)));
	// copy from host to device
	HANDLE_ERROR(hipMemcpy(d_transform0, &t0, sizeof(transf), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_transform1, &t1, sizeof(transf), hipMemcpyHostToDevice));

	const unsigned int block_size = 16;
	dim3 threads;
	dim3 grids;

	// ====================================== check bounding sphere ======================================

	// prepare result array
	thrust::device_vector<bool> d_vertex0_preprocess_result(_num_vtx);
	thrust::device_vector<bool> d_vertex1_preprocess_result(other->_num_vtx);

	// bounding sphere transformation
	BoundingSphere sphere0(this->m_bounding_sphere), sphere1(other->m_bounding_sphere);
	sphere0.center = t0.getVertex(sphere0.center);
	sphere1.center = t1.getVertex(sphere1.center);

	BoundingSphere* d_sphere0;
	BoundingSphere* d_sphere1;
	// allocate memory
	HANDLE_ERROR(hipMalloc((void**)&d_sphere0, sizeof(BoundingSphere)));
	HANDLE_ERROR(hipMalloc((void**)&d_sphere1, sizeof(BoundingSphere)));
	// copy from host to device
	HANDLE_ERROR(hipMemcpy(d_sphere0, &sphere0, sizeof(BoundingSphere), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_sphere1, &sphere1, sizeof(BoundingSphere), hipMemcpyHostToDevice));

	// =================================== check mesh0 with sphere1 =============================
	// check mesh0 intersect with sphere1, culling vertex
	threads = dim3(block_size);
	grids = dim3((d_vertex0_preprocess_result.size() + (block_size - 1)) / block_size);
	MeshPreprocessCUDA << < grids, threads >> > (
		thrust::raw_pointer_cast(d_mesh0_vtxs.data()),
		thrust::raw_pointer_cast(d_vertex0_preprocess_result.data()),
		d_sphere1,
		d_transform0,
		d_vertex0_preprocess_result.size()
		);
	// culling triangles
	thrust::device_vector<bool> d_tri0_culling_stencil(_num_tri);
	threads = dim3(block_size);
	grids = dim3((d_tri0_culling_stencil.size() + (block_size - 1)) / block_size);
	TriCullingCUDA << < grids, threads >> > (
		thrust::raw_pointer_cast(d_mesh0_tris.data()),
		thrust::raw_pointer_cast(d_tri0_culling_stencil.data()),
		thrust::raw_pointer_cast(d_vertex0_preprocess_result.data()),
		d_tri0_culling_stencil.size()
		);
	// reduce triangle id
	thrust::counting_iterator<int> counting(0);
	thrust::device_vector<int> d_culled_tri0_ids(_num_tri);
	auto end0 = thrust::copy_if(
		counting,
		counting + _num_tri,
		d_tri0_culling_stencil.begin(),
		d_culled_tri0_ids.begin(),
		thrust::identity<bool>()
	);
	d_culled_tri0_ids.resize(end0 - d_culled_tri0_ids.begin());

	// =================================== check mesh1 with sphere0 =============================
	// check mesh1 intersect with sphere0, culling vertex
	grids = dim3((d_vertex1_preprocess_result.size() + (block_size - 1)) / block_size);
	MeshPreprocessCUDA << < grids, threads >> > (
		thrust::raw_pointer_cast(d_mesh1_vtxs.data()),
		thrust::raw_pointer_cast(d_vertex1_preprocess_result.data()),
		d_sphere0,
		d_transform1,
		d_vertex1_preprocess_result.size()
		);
	// culling mesh1 triangles
	thrust::device_vector<bool> d_tri1_culling_stencil(other->_num_tri);
	threads = dim3(block_size);
	grids = dim3((d_tri1_culling_stencil.size() + (block_size - 1)) / block_size);
	TriCullingCUDA << < grids, threads >> > (
		thrust::raw_pointer_cast(d_mesh1_tris.data()),
		thrust::raw_pointer_cast(d_tri1_culling_stencil.data()),
		thrust::raw_pointer_cast(d_vertex1_preprocess_result.data()),
		d_tri1_culling_stencil.size()
		);
	// reduce triangle id
	thrust::device_vector<int> d_culled_tri1_ids(other->_num_tri);
	auto end1 = thrust::copy_if(
		counting,
		counting + other->_num_tri,
		d_tri1_culling_stencil.begin(),
		d_culled_tri1_ids.begin(),
		thrust::identity<bool>()
	);
	d_culled_tri1_ids.resize(end1 - d_culled_tri1_ids.begin());
	
	printf("mesh0 tri num: %d   mesh1 tri num: %d\n", d_culled_tri0_ids.size(), d_culled_tri1_ids.size());


	// ====================================== use cuda intersect ===========================================
	
	// call kernel
	threads = dim3(block_size, block_size);
	grids = dim3(
		(d_culled_tri0_ids.size() + (block_size - 1)) / block_size,
		(d_culled_tri1_ids.size() + (block_size - 1)) / block_size);
	//grids = dim3(500, 500);
	MeshIntersectCUDA << < grids, threads >> > (
		thrust::raw_pointer_cast(d_culled_tri0_ids.data()), thrust::raw_pointer_cast(d_culled_tri1_ids.data()),
		thrust::raw_pointer_cast(d_mesh0_vtxs.data()), thrust::raw_pointer_cast(d_mesh0_tris.data()), 
		thrust::raw_pointer_cast(d_mesh1_vtxs.data()), thrust::raw_pointer_cast(d_mesh1_tris.data()), 
		d_culled_tri0_ids.size(), d_culled_tri1_ids.size(),
		thrust::raw_pointer_cast(d_triangle0_result.data()), thrust::raw_pointer_cast(d_triangle1_result.data()));
	hipDeviceSynchronize();

	// copy result from device to host
	thrust::host_vector<bool> h_triangle0_result = d_triangle0_result;
	thrust::host_vector<bool> h_triangle1_result = d_triangle1_result;

	

	int mesh0_collide_num = 0;
	int mesh1_collide_num = 0;
	for (int i = 0; i < _num_tri; i++)
	{
		if (h_triangle0_result[i]) mesh0_collide_num++;
		if (h_triangle1_result[i]) mesh1_collide_num++;
	}
	printf("mesh0: %d, mesh1: %d\n", mesh0_collide_num, mesh1_collide_num);

	int mesh0_first_tri = -1;
	for (int i = 0; i < _num_tri; i++)
	{
		if (h_triangle0_result[i]) {
			mesh0_first_tri = i;
			break;
		}
	}
	if (mesh0_first_tri == -1)
	{
		return;
	}

	int mesh1_first_tri = -1;
	for (int i = 0; i < _num_tri; i++)
	{
		if (h_triangle1_result[i]) {
			mesh1_first_tri = i;
			break;
		}
	}
	if (mesh1_first_tri == -1)
	{
		return;
	}

	for (int i = 0; i < _num_tri; i++)
	{
		if (h_triangle0_result[i])
		{
			rets.push_back(id_pair(i, mesh1_first_tri, false));
		}
		if (h_triangle1_result[i])
		{
			rets.push_back(id_pair(mesh0_first_tri, i, false));
		}
	}

	// free memory
	HANDLE_ERROR(hipFree(d_transform0));
	HANDLE_ERROR(hipFree(d_transform1));
	HANDLE_ERROR(hipFree(d_sphere0));
	HANDLE_ERROR(hipFree(d_sphere1));
#endif // !GPU_ACCELE
}
